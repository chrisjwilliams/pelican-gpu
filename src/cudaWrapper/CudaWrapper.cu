// This files provides a wrapper for all CUDA calls
#include "cutil_inline.h"
#include "CudaWrapper.h"

// =================== CUDA-related function implementation ===========================

// Memory allocation calls
extern "C" void allocateGpuMemory(void **buffer, unsigned long size)
{    cutilSafeCall( hipMalloc(buffer, size));    }

extern "C" void freeGpuMemory(void *buffer)
{    cutilSafeCall(hipFree(buffer));    }

// Memory management calls
extern "C" void copyData(void *destination, void *source, unsigned long size, CopyDirection direction)
{    
    switch(direction) {
        case 1: 
        {
            cutilSafeCall( hipMemcpy(destination, source, size, hipMemcpyHostToDevice) );              
            break;
        }
        case 2:
        {
            cutilSafeCall( hipMemcpy(destination, source, size, hipMemcpyDeviceToHost) );              
            break;
        }
        case 3:
        {
            cutilSafeCall( hipMemcpy(destination, source, size, hipMemcpyDeviceToDevice) );              
            break;
        }
    }
}

extern "C" void setGpuBuffer(void *buffer, char value, unsigned long size)
{    hipMemset(buffer, value, size);    }

extern "C" void copyDataToSymbol(void *symbol, void *source, unsigned long size)
{    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(symbol), source, size) );    }

// GPU management calls
extern "C"  unsigned getDeviceInformation(DeviceInfo** devices)
{    return NULL;    }

extern "C" void setDevice(unsigned deviceId)
{    cutilSafeCall( hipSetDevice(deviceId));    }
